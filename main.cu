#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include "CrankNicolsonSOR.cuh"
#include <iostream>
#include <Windows.h>
#include <vector>

using namespace std;

int main()
{
	
	hipDeviceProp_t data;
	hipGetDeviceProperties(&data, 0);
	cout << "Device name: " << data.name << endl;
	cout << data.maxThreadsPerBlock << " - MAX THREADS PER BLOCK" << endl;
	cout << data.sharedMemPerBlock << " - SHARED MEM PER BLOCK" << endl;
	

	double s0_ = 100.0, r_ = 0.05, sigma_ = 0.2;
	double K_ = 100.0, T_ = 1., sl_ = 0.0, su_ = 200.0, omega = 1.;
	const int imax = 500, jmax = 500, nmax = 50;

	BSPDE PDE(s0_, r_, sigma_, K_, T_, sl_, su_, imax, jmax);

	double V[jmax + 1];

	for (int j = 0; j <= jmax; j++)
	{
		V[j] = PDE.f_h(j);
	}

	double *d_V;
	BSPDE *d_PDE;

	hipMalloc(&d_PDE, sizeof(PDE));
	hipMemcpy(d_PDE, &PDE, sizeof(PDE), hipMemcpyHostToDevice);

	double *data;
	hipMalloc(&data, sizeof(double) * imax*(jmax+1));

	hipMalloc(&d_V, sizeof(V));
	hipMemcpy(d_V, &V, sizeof(V), hipMemcpyHostToDevice);

	SOR <<<1, jmax - 1 >>>(d_V, d_PDE, omega, nmax, data);

	double *dataHost;
	dataHost = new double [imax*(jmax + 1)];
	hipMemcpy(dataHost, data, sizeof(double) * imax*(jmax + 1), hipMemcpyDeviceToHost);

	vector<vector<double>> dataVector;
	dataVector.resize(imax + 1);
	for (int i = 0; i <= imax;i++)
		dataVector[i].resize(jmax + 1);
	for (int j = 0; j <= jmax;j++)
		dataVector[imax][j] = PDE.f_h(j);
	for (int i = 0; i < imax; i++)
		for (int j = 0; j <= jmax; j++)
			dataVector[i][j] = dataHost[i*jmax + j];

	delete [] dataHost;
	hipMemcpy(&V, d_V, sizeof(V), hipMemcpyDeviceToHost);
	
	double price;
	double *d_price;

	hipMalloc(&d_price, sizeof(double));
	
	Price<<<1, 1>>>(d_price, d_V, d_PDE, s0_);

	hipMemcpy(&price, d_price, sizeof(double), hipMemcpyDeviceToHost);
	cout << "Price: " << price << endl << endl;

	hipFree(d_V);
	hipFree(d_PDE);
	hipFree(d_price);

	system("pause");
	return 0;
}